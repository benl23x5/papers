
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include "sdk-kernel.cu"

#define cutilSafeCallNoSync(err)     __cudaSafeCallNoSync(err, __FILE__, __LINE__)

inline void __cudaSafeCallNoSync( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : cudaSafeCallNoSync() Runtime API error : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}


extern "C"
void blackscholes
(
    float               *h_CallResult,
    float               *h_PutResult,
    float               *d_StockPrice,
    float               *d_OptionStrike,
    float               *d_OptionYears,
    const float         riskfree,
    const float         volatility,
    const int           opt_n
)
{
    const float opt_sz = opt_n * sizeof(float);
    float * d_CallResult;
    float * d_PutResult;

    /*
     * Allocate device memory
     */
    cutilSafeCallNoSync( hipMalloc((void **)&d_CallResult,   opt_sz) );
    cutilSafeCallNoSync( hipMalloc((void **)&d_PutResult,    opt_sz) );

    /*
     * Execute
     */
    BlackScholesGPU<<<480, 128>>>(
        d_CallResult,
        d_PutResult,
        d_StockPrice,
        d_OptionStrike,
        d_OptionYears,
        riskfree,
        volatility,
        opt_n
    );
    cutilSafeCallNoSync( hipDeviceSynchronize() );

    /*
     * Copy result back to host
     */
//    cutilSafeCallNoSync( hipMemcpy(h_CallResult, d_CallResult, opt_sz, hipMemcpyDeviceToHost) );
//    cutilSafeCallNoSync( hipMemcpy(h_PutResult,  d_PutResult,  opt_sz, hipMemcpyDeviceToHost) );

    /*
     * Release device memory
     */
    cutilSafeCallNoSync( hipFree(d_PutResult)    );
    cutilSafeCallNoSync( hipFree(d_CallResult)   );
}

