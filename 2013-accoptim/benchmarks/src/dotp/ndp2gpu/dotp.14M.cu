#include "hip/hip_runtime.h"
#include "config.h"
#include "vcode.h"
#include <cvl.h>
#include "y.tab.h"
#include <cutil_inline.h>
#include "defins.cuh"

MAXALIGN *ComputeMemory = NULL;

extern "C" void init (MAXALIGN *mem) {
  ComputeMemory = mem;
}

__global__ void fused0Kernel(MAXALIGN *data, int dst, int s0, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    int *pDst = (int*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    
    pDst[address] = (100);
  }
}

__global__ void fused1Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    
    pDst[address] = (divide(pSrc0[address], (z_to_d(pSrc1[address]))));
  }
}

__global__ void fused2Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    float *pSrc2 = (float*)(&data[s2]);
    
    pDst[address] = (times((divide(pSrc0[address], (z_to_d(pSrc1[address])))), pSrc2[address]));
  }
}

void fused0(vec_p d, vec_p s0, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused0Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, len, scratch);
  cutilCheckMsg("fused0 execution failed\n");
}

void fused1(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused1Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused1 execution failed\n");
}

void fused2(vec_p d, vec_p s0, vec_p s1, vec_p s2, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused2Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, len, scratch);
  cutilCheckMsg("fused2 execution failed\n");
}

make_no_scratch(fused0)
make_no_scratch(fused1)
make_no_scratch(fused2)
make_inplace(fused0, INPLACE_NONE)
make_inplace(fused1, INPLACE_1)
make_inplace(fused2, INPLACE_1)
vopdes_t vops[] = {
  {FUSED, "fused0", 1, 1,
  {Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Int,},
  {COMPAT1,},
  {1,},
  Elwise1},
  {FUSED, "fused1", 2, 1,
  {Float,Int,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise2},
  {FUSED, "fused2", 3, 1,
  {Float,Int,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise3},
  };

cvl_triple_t cvl_funs[] = {
  { { (void (*)())fused0, (int (*)())fused0_scratch, (unsigned (*)())fused0_inplace },},
  { { (void (*)())fused1, (int (*)())fused1_scratch, (unsigned (*)())fused1_inplace },},
  { { (void (*)())fused2, (int (*)())fused2_scratch, (unsigned (*)())fused2_inplace },},
  };
/*
fused OP0#2 ($0 : SEGDES) = (DIST INT @ (CONST INT 100) $0)
fused OP1#1 ($0 : FLOAT, $1 : INT) = (/ FLOAT @ $0 (I_TO_F @ $1))
fused OP2#1 ($0 : FLOAT, $1 : INT, $2 : FLOAT) = (* FLOAT @ (/ FLOAT @ $0 (I_TO_F @ $1)) $2)
*/
