#include "hip/hip_runtime.h"
#include "config.h"
#include "vcode.h"
#include <cvl.h>
#include "y.tab.h"
#include <cutil_inline.h>
#include "defins.cuh"

MAXALIGN *ComputeMemory = NULL;

extern "C" void init (MAXALIGN *mem) {
  ComputeMemory = mem;
}

__global__ void fused0Kernel(MAXALIGN *data, int dst, int s0, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    
    pDst[address] = (100.0);
  }
}

__global__ void fused1Kernel(MAXALIGN *data, int dst, int s0, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    
    pDst[address] = (1.0);
  }
}

__global__ void fused2Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    int *pSrc2 = (int*)(&data[s2]);
    
    pDst[address] = (times(pSrc0[address], (divide((z_to_d(pSrc1[address])), (z_to_d(pSrc2[address]))))));
  }
}

__global__ void fused3Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    
    pDst[address] = (divide(pSrc0[address], (z_to_d(pSrc1[address]))));
  }
}

__global__ void fused4Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int s4, int s5, int s6, int s7, int s8, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    float *pSrc2 = (float*)(&data[s2]);
    float *pSrc3 = (float*)(&data[s3]);
    float *pSrc4 = (float*)(&data[s4]);
    int *pSrc5 = (int*)(&data[s5]);
    float *pSrc6 = (float*)(&data[s6]);
    float *pSrc7 = (float*)(&data[s7]);
    float *pSrc8 = (float*)(&data[s8]);
    
    pDst[address] = (divide((plus((log((divide(pSrc0[address], pSrc1[address])))), (times((plus(pSrc2[address], (divide((times(pSrc3[address], pSrc4[address])), (2.0))))), pSrc6[address])))), (times(pSrc7[address], (sqrt(pSrc8[address]))))));
  }
}

__global__ void fused5Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int s4, int s5, int s6, int s7, int s8, int s9, int s10, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    float *pSrc2 = (float*)(&data[s2]);
    float *pSrc3 = (float*)(&data[s3]);
    float *pSrc4 = (float*)(&data[s4]);
    int *pSrc5 = (int*)(&data[s5]);
    float *pSrc6 = (float*)(&data[s6]);
    float *pSrc7 = (float*)(&data[s7]);
    float *pSrc8 = (float*)(&data[s8]);
    float *pSrc9 = (float*)(&data[s9]);
    float *pSrc10 = (float*)(&data[s10]);
    
    pDst[address] = (minus((divide((plus((log((divide(pSrc0[address], pSrc1[address])))), (times((plus(pSrc2[address], (divide((times(pSrc3[address], pSrc4[address])), (2.0))))), pSrc6[address])))), (times(pSrc7[address], (sqrt(pSrc8[address])))))), (times(pSrc9[address], (sqrt(pSrc10[address]))))));
  }
}

__global__ void fused6Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int s4, int s5, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    float *pSrc2 = (float*)(&data[s2]);
    float *pSrc3 = (float*)(&data[s3]);
    float *pSrc4 = (float*)(&data[s4]);
    float *pSrc5 = (float*)(&data[s5]);
    
    pDst[address] = (minus((times(pSrc0[address], pSrc1[address])), (times((times(pSrc2[address], (exp((times(pSrc3[address], pSrc4[address])))))), pSrc5[address]))));
  }
}

__global__ void fused7Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int s4, int s5, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    float *pSrc2 = (float*)(&data[s2]);
    float *pSrc3 = (float*)(&data[s3]);
    float *pSrc4 = (float*)(&data[s4]);
    float *pSrc5 = (float*)(&data[s5]);
    
    pDst[address] = (minus((times((times(pSrc0[address], (exp((times(pSrc1[address], pSrc2[address])))))), pSrc3[address])), (times(pSrc4[address], pSrc5[address]))));
  }
}

__global__ void fused8Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    
    pDst[address] = (minus((0.0), pSrc1[address]));
  }
}

__global__ void fused9Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    
    pDst[address] = (times(pSrc0[address], (atan(pSrc1[address]))));
  }
}

__global__ void fused10Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    float *pSrc2 = (float*)(&data[s2]);
    float *pSrc3 = (float*)(&data[s3]);
    
    pDst[address] = (selection((gt(pSrc0[address], (0.0))), pSrc2[address], pSrc3[address]));
  }
}

__global__ void fused11Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int s4, int s5, int s6, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    int *pSrc2 = (int*)(&data[s2]);
    float *pSrc3 = (float*)(&data[s3]);
    int *pSrc4 = (int*)(&data[s4]);
    float *pSrc5 = (float*)(&data[s5]);
    float *pSrc6 = (float*)(&data[s6]);
    
    pDst[address] = (divide((1.0), (plus((1.0), (times((0.2316419), (selection((gt(pSrc3[address], (0.0))), pSrc5[address], pSrc6[address]))))))));
  }
}

__global__ void fused12Kernel(MAXALIGN *data, int dst, int s0, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    int *pDst = (int*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    
    pDst[address] = (3);
  }
}

__global__ void fused13Kernel(MAXALIGN *data, int dst, int s0, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    int *pDst = (int*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    
    pDst[address] = (4);
  }
}

__global__ void fused14Kernel(MAXALIGN *data, int dst, int s0, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    int *pDst = (int*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    
    pDst[address] = (5);
  }
}

__global__ void fused15Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int s4, int s5, int s6, int s7, int s8, int s9, int s10, int s11, int s12, int s13, int s14, int s15, int s16, int s17, int s18, int s19, int s20, int s21, int s22, int s23, int s24, int s25, int s26, int s27, int s28, int s29, int s30, int s31, int s32, int s33, int s34, int s35, int s36, int s37, int s38, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    int *pSrc2 = (int*)(&data[s2]);
    float *pSrc3 = (float*)(&data[s3]);
    float *pSrc4 = (float*)(&data[s4]);
    float *pSrc5 = (float*)(&data[s5]);
    int *pSrc6 = (int*)(&data[s6]);
    float *pSrc7 = (float*)(&data[s7]);
    float *pSrc8 = (float*)(&data[s8]);
    int *pSrc9 = (int*)(&data[s9]);
    int *pSrc10 = (int*)(&data[s10]);
    int *pSrc11 = (int*)(&data[s11]);
    int *pSrc12 = (int*)(&data[s12]);
    int *pSrc13 = (int*)(&data[s13]);
    float *pSrc14 = (float*)(&data[s14]);
    int *pSrc15 = (int*)(&data[s15]);
    float *pSrc16 = (float*)(&data[s16]);
    float *pSrc17 = (float*)(&data[s17]);
    float *pSrc18 = (float*)(&data[s18]);
    int *pSrc19 = (int*)(&data[s19]);
    int *pSrc20 = (int*)(&data[s20]);
    int *pSrc21 = (int*)(&data[s21]);
    float *pSrc22 = (float*)(&data[s22]);
    int *pSrc23 = (int*)(&data[s23]);
    float *pSrc24 = (float*)(&data[s24]);
    float *pSrc25 = (float*)(&data[s25]);
    int *pSrc26 = (int*)(&data[s26]);
    int *pSrc27 = (int*)(&data[s27]);
    int *pSrc28 = (int*)(&data[s28]);
    float *pSrc29 = (float*)(&data[s29]);
    int *pSrc30 = (int*)(&data[s30]);
    float *pSrc31 = (float*)(&data[s31]);
    float *pSrc32 = (float*)(&data[s32]);
    int *pSrc33 = (int*)(&data[s33]);
    float *pSrc34 = (float*)(&data[s34]);
    float *pSrc35 = (float*)(&data[s35]);
    float *pSrc36 = (float*)(&data[s36]);
    int *pSrc37 = (int*)(&data[s37]);
    float *pSrc38 = (float*)(&data[s38]);
    
    pDst[address] = (minus((1.0), (times((times((divide((1.0), (sqrt((times((2.0), pSrc3[address])))))), (exp((divide((times(pSrc4[address], (selection((gt(pSrc5[address], (0.0))), pSrc7[address], pSrc8[address])))), (2.0))))))), (plus((plus((plus((plus((times((0.31938153), (divide((1.0), (plus((1.0), (times((0.2316419), (selection((gt(pSrc14[address], (0.0))), pSrc16[address], pSrc17[address])))))))))), (times((times(pSrc18[address], (divide((1.0), (plus((1.0), (times((0.2316419), (selection((gt(pSrc22[address], (0.0))), pSrc24[address], pSrc25[address])))))))))), (divide((1.0), (plus((1.0), (times((0.2316419), (selection((gt(pSrc29[address], (0.0))), pSrc31[address], pSrc32[address])))))))))))), (times((1.781477937), pSrc34[address])))), (times(pSrc35[address], pSrc36[address])))), (times((1.330274429), pSrc38[address]))))))));
  }
}

__global__ void fused16Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    int *pDst = (int*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    
    pDst[address] = (lt(pSrc0[address], (0.0)));
  }
}

__global__ void fused17Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    int *pDst = (int*)(&data[dst]);
    float *pSrc0 = (float*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    
    pDst[address] = (b_to_z((lt(pSrc0[address], (0.0)))));
  }
}

__global__ void fused18Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int s3, int s4, int s5, int s6, int s7, int s8, int s9, int s10, int s11, int s12, int s13, int s14, int s15, int s16, int s17, int s18, int s19, int s20, int s21, int s22, int s23, int s24, int s25, int s26, int s27, int s28, int s29, int s30, int s31, int s32, int s33, int s34, int s35, int s36, int s37, int s38, int s39, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    int *pSrc2 = (int*)(&data[s2]);
    int *pSrc3 = (int*)(&data[s3]);
    float *pSrc4 = (float*)(&data[s4]);
    float *pSrc5 = (float*)(&data[s5]);
    float *pSrc6 = (float*)(&data[s6]);
    int *pSrc7 = (int*)(&data[s7]);
    float *pSrc8 = (float*)(&data[s8]);
    float *pSrc9 = (float*)(&data[s9]);
    int *pSrc10 = (int*)(&data[s10]);
    int *pSrc11 = (int*)(&data[s11]);
    int *pSrc12 = (int*)(&data[s12]);
    int *pSrc13 = (int*)(&data[s13]);
    int *pSrc14 = (int*)(&data[s14]);
    float *pSrc15 = (float*)(&data[s15]);
    int *pSrc16 = (int*)(&data[s16]);
    float *pSrc17 = (float*)(&data[s17]);
    float *pSrc18 = (float*)(&data[s18]);
    float *pSrc19 = (float*)(&data[s19]);
    int *pSrc20 = (int*)(&data[s20]);
    int *pSrc21 = (int*)(&data[s21]);
    int *pSrc22 = (int*)(&data[s22]);
    float *pSrc23 = (float*)(&data[s23]);
    int *pSrc24 = (int*)(&data[s24]);
    float *pSrc25 = (float*)(&data[s25]);
    float *pSrc26 = (float*)(&data[s26]);
    int *pSrc27 = (int*)(&data[s27]);
    int *pSrc28 = (int*)(&data[s28]);
    int *pSrc29 = (int*)(&data[s29]);
    float *pSrc30 = (float*)(&data[s30]);
    int *pSrc31 = (int*)(&data[s31]);
    float *pSrc32 = (float*)(&data[s32]);
    float *pSrc33 = (float*)(&data[s33]);
    int *pSrc34 = (int*)(&data[s34]);
    float *pSrc35 = (float*)(&data[s35]);
    float *pSrc36 = (float*)(&data[s36]);
    float *pSrc37 = (float*)(&data[s37]);
    int *pSrc38 = (int*)(&data[s38]);
    float *pSrc39 = (float*)(&data[s39]);
    
    pDst[address] = (minus((1.0), (minus((1.0), (times((times((divide((1.0), (sqrt((times((2.0), pSrc4[address])))))), (exp((divide((times(pSrc5[address], (selection((gt(pSrc6[address], (0.0))), pSrc8[address], pSrc9[address])))), (2.0))))))), (plus((plus((plus((plus((times((0.31938153), (divide((1.0), (plus((1.0), (times((0.2316419), (selection((gt(pSrc15[address], (0.0))), pSrc17[address], pSrc18[address])))))))))), (times((times(pSrc19[address], (divide((1.0), (plus((1.0), (times((0.2316419), (selection((gt(pSrc23[address], (0.0))), pSrc25[address], pSrc26[address])))))))))), (divide((1.0), (plus((1.0), (times((0.2316419), (selection((gt(pSrc30[address], (0.0))), pSrc32[address], pSrc33[address])))))))))))), (times((1.781477937), pSrc35[address])))), (times(pSrc36[address], pSrc37[address])))), (times((1.330274429), pSrc39[address]))))))))));
  }
}

__global__ void fused19Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    int *pDst = (int*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    int *pSrc1 = (int*)(&data[s1]);
    int *pSrc2 = (int*)(&data[s2]);
    
    pDst[address] = (eq((minus(pSrc0[address], pSrc1[address])), pSrc2[address]));
  }
}

__global__ void fused20Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    
    pDst[address] = (minus((1.0), pSrc1[address]));
  }
}

__global__ void fused21Kernel(MAXALIGN *data, int dst, int s0, int s1, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  if (address < len) {
    float *pDst = (float*)(&data[dst]);
    int *pSrc0 = (int*)(&data[s0]);
    float *pSrc1 = (float*)(&data[s1]);
    
    pDst[address] = (times((exp((z_to_d(pSrc0[address])))), (log(pSrc1[address]))));
  }
}

void fused0(vec_p d, vec_p s0, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused0Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, len, scratch);
  cutilCheckMsg("fused0 execution failed\n");
}

void fused1(vec_p d, vec_p s0, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused1Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, len, scratch);
  cutilCheckMsg("fused1 execution failed\n");
}

void fused2(vec_p d, vec_p s0, vec_p s1, vec_p s2, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused2Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, len, scratch);
  cutilCheckMsg("fused2 execution failed\n");
}

void fused3(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused3Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused3 execution failed\n");
}

void fused4(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, vec_p s4, vec_p s5, vec_p s6, vec_p s7, vec_p s8, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused4Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, s4, s5, s6, s7, s8, len, scratch);
  cutilCheckMsg("fused4 execution failed\n");
}

void fused5(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, vec_p s4, vec_p s5, vec_p s6, vec_p s7, vec_p s8, vec_p s9, vec_p s10, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused5Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, s4, s5, s6, s7, s8, s9, s10, len, scratch);
  cutilCheckMsg("fused5 execution failed\n");
}

void fused6(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, vec_p s4, vec_p s5, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused6Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, s4, s5, len, scratch);
  cutilCheckMsg("fused6 execution failed\n");
}

void fused7(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, vec_p s4, vec_p s5, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused7Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, s4, s5, len, scratch);
  cutilCheckMsg("fused7 execution failed\n");
}

void fused8(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused8Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused8 execution failed\n");
}

void fused9(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused9Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused9 execution failed\n");
}

void fused10(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused10Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, len, scratch);
  cutilCheckMsg("fused10 execution failed\n");
}

void fused11(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, vec_p s4, vec_p s5, vec_p s6, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused11Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, s4, s5, s6, len, scratch);
  cutilCheckMsg("fused11 execution failed\n");
}

void fused12(vec_p d, vec_p s0, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused12Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, len, scratch);
  cutilCheckMsg("fused12 execution failed\n");
}

void fused13(vec_p d, vec_p s0, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused13Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, len, scratch);
  cutilCheckMsg("fused13 execution failed\n");
}

void fused14(vec_p d, vec_p s0, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused14Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, len, scratch);
  cutilCheckMsg("fused14 execution failed\n");
}

void fused15(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, vec_p s4, vec_p s5, vec_p s6, vec_p s7, vec_p s8, vec_p s9, vec_p s10, vec_p s11, vec_p s12, vec_p s13, vec_p s14, vec_p s15, vec_p s16, vec_p s17, vec_p s18, vec_p s19, vec_p s20, vec_p s21, vec_p s22, vec_p s23, vec_p s24, vec_p s25, vec_p s26, vec_p s27, vec_p s28, vec_p s29, vec_p s30, vec_p s31, vec_p s32, vec_p s33, vec_p s34, vec_p s35, vec_p s36, vec_p s37, vec_p s38, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused15Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, s4, s5, s6, s7, s8, s9, s10, s11, s12, s13, s14, s15, s16, s17, s18, s19, s20, s21, s22, s23, s24, s25, s26, s27, s28, s29, s30, s31, s32, s33, s34, s35, s36, s37, s38, len, scratch);
  cutilCheckMsg("fused15 execution failed\n");
}

void fused16(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused16Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused16 execution failed\n");
}

void fused17(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused17Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused17 execution failed\n");
}

void fused18(vec_p d, vec_p s0, vec_p s1, vec_p s2, vec_p s3, vec_p s4, vec_p s5, vec_p s6, vec_p s7, vec_p s8, vec_p s9, vec_p s10, vec_p s11, vec_p s12, vec_p s13, vec_p s14, vec_p s15, vec_p s16, vec_p s17, vec_p s18, vec_p s19, vec_p s20, vec_p s21, vec_p s22, vec_p s23, vec_p s24, vec_p s25, vec_p s26, vec_p s27, vec_p s28, vec_p s29, vec_p s30, vec_p s31, vec_p s32, vec_p s33, vec_p s34, vec_p s35, vec_p s36, vec_p s37, vec_p s38, vec_p s39, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused18Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, s3, s4, s5, s6, s7, s8, s9, s10, s11, s12, s13, s14, s15, s16, s17, s18, s19, s20, s21, s22, s23, s24, s25, s26, s27, s28, s29, s30, s31, s32, s33, s34, s35, s36, s37, s38, s39, len, scratch);
  cutilCheckMsg("fused18 execution failed\n");
}

void fused19(vec_p d, vec_p s0, vec_p s1, vec_p s2, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused19Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, len, scratch);
  cutilCheckMsg("fused19 execution failed\n");
}

void fused20(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused20Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused20 execution failed\n");
}

void fused21(vec_p d, vec_p s0, vec_p s1, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused21Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, len, scratch);
  cutilCheckMsg("fused21 execution failed\n");
}

make_no_scratch(fused0)
make_no_scratch(fused1)
make_no_scratch(fused2)
make_no_scratch(fused3)
make_no_scratch(fused4)
make_no_scratch(fused5)
make_no_scratch(fused6)
make_no_scratch(fused7)
make_no_scratch(fused8)
make_no_scratch(fused9)
make_no_scratch(fused10)
make_no_scratch(fused11)
make_no_scratch(fused12)
make_no_scratch(fused13)
make_no_scratch(fused14)
make_no_scratch(fused15)
make_no_scratch(fused16)
make_no_scratch(fused17)
make_no_scratch(fused18)
make_no_scratch(fused19)
make_no_scratch(fused20)
make_no_scratch(fused21)
make_inplace(fused0, INPLACE_NONE)
make_inplace(fused1, INPLACE_NONE)
make_inplace(fused2, INPLACE_1)
make_inplace(fused3, INPLACE_1)
make_inplace(fused4, INPLACE_1)
make_inplace(fused5, INPLACE_1)
make_inplace(fused6, INPLACE_1)
make_inplace(fused7, INPLACE_1)
make_inplace(fused8, INPLACE_2)
make_inplace(fused9, INPLACE_1)
make_inplace(fused10, INPLACE_1)
make_inplace(fused11, INPLACE_4)
make_inplace(fused12, INPLACE_NONE)
make_inplace(fused13, INPLACE_NONE)
make_inplace(fused14, INPLACE_NONE)
make_inplace(fused15, INPLACE_4)
make_inplace(fused16, INPLACE_1)
make_inplace(fused17, INPLACE_1)
make_inplace(fused18, INPLACE_5)
make_inplace(fused19, INPLACE_1)
make_inplace(fused20, INPLACE_2)
make_inplace(fused21, INPLACE_1)
vopdes_t vops[] = {
  {FUSED, "fused0", 1, 1,
  {Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {COMPAT1,},
  {1,},
  Elwise1},
  {FUSED, "fused1", 1, 1,
  {Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {COMPAT1,},
  {1,},
  Elwise1},
  {FUSED, "fused2", 3, 1,
  {Float,Int,Int,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise3},
  {FUSED, "fused3", 2, 1,
  {Float,Int,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise2},
  {FUSED, "fused4", 9, 1,
  {Float,Float,Float,Float,Float,Segdes,Float,Float,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise9},
  {FUSED, "fused5", 11, 1,
  {Float,Float,Float,Float,Float,Segdes,Float,Float,Float,Float,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise11},
  {FUSED, "fused6", 6, 1,
  {Float,Float,Float,Float,Float,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise6},
  {FUSED, "fused7", 6, 1,
  {Float,Float,Float,Float,Float,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise6},
  {FUSED, "fused8", 2, 1,
  {Segdes,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {COMPAT1,},
  {1,},
  Elwise2},
  {FUSED, "fused9", 2, 1,
  {Float,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise2},
  {FUSED, "fused10", 4, 1,
  {Float,Segdes,Float,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise4},
  {FUSED, "fused11", 7, 1,
  {Segdes,Segdes,Segdes,Float,Segdes,Float,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,NONE,NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {COMPAT1,},
  {1,},
  Elwise7},
  {FUSED, "fused12", 1, 1,
  {Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Int,},
  {COMPAT1,},
  {1,},
  Elwise1},
  {FUSED, "fused13", 1, 1,
  {Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Int,},
  {COMPAT1,},
  {1,},
  Elwise1},
  {FUSED, "fused14", 1, 1,
  {Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Int,},
  {COMPAT1,},
  {1,},
  Elwise1},
  {FUSED, "fused15", 39, 1,
  {Segdes,Segdes,Segdes,Float,Float,Float,Segdes,Float,Float,Segdes,Segdes,Segdes,Segdes,Segdes,Float,Segdes,Float,Float,Float,Segdes,Segdes,Segdes,Float,Segdes,Float,Float,Segdes,Segdes,Segdes,Float,Segdes,Float,Float,Segdes,Float,Float,Float,Segdes,Float,Illegal},
  {NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,Illegal},
  {Float,},
  {COMPAT1,},
  {1,},
  Elwise39},
  {FUSED, "fused16", 2, 1,
  {Float,Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Bool,},
  {AGREE1,},
  {1,},
  Elwise2},
  {FUSED, "fused17", 2, 1,
  {Float,Segdes,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Int,},
  {AGREE1,},
  {1,},
  Elwise2},
  {FUSED, "fused18", 40, 1,
  {Segdes,Segdes,Segdes,Segdes,Float,Float,Float,Segdes,Float,Float,Segdes,Segdes,Segdes,Segdes,Segdes,Float,Segdes,Float,Float,Float,Segdes,Segdes,Segdes,Float,Segdes,Float,Float,Segdes,Segdes,Segdes,Float,Segdes,Float,Float,Segdes,Float,Float,Float,Segdes,Float},
  {NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE,NONE},
  {Float,},
  {COMPAT1,},
  {1,},
  Elwise40},
  {FUSED, "fused19", 3, 1,
  {Int,Int,Int,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Bool,},
  {AGREE1,},
  {1,},
  Elwise3},
  {FUSED, "fused20", 2, 1,
  {Segdes,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {COMPAT1,},
  {1,},
  Elwise2},
  {FUSED, "fused21", 2, 1,
  {Int,Float,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Float,},
  {AGREE1,},
  {1,},
  Elwise2},
  };

cvl_triple_t cvl_funs[] = {
  { { (void (*)())fused0, (int (*)())fused0_scratch, (unsigned (*)())fused0_inplace },},
  { { (void (*)())fused1, (int (*)())fused1_scratch, (unsigned (*)())fused1_inplace },},
  { { (void (*)())fused2, (int (*)())fused2_scratch, (unsigned (*)())fused2_inplace },},
  { { (void (*)())fused3, (int (*)())fused3_scratch, (unsigned (*)())fused3_inplace },},
  { { (void (*)())fused4, (int (*)())fused4_scratch, (unsigned (*)())fused4_inplace },},
  { { (void (*)())fused5, (int (*)())fused5_scratch, (unsigned (*)())fused5_inplace },},
  { { (void (*)())fused6, (int (*)())fused6_scratch, (unsigned (*)())fused6_inplace },},
  { { (void (*)())fused7, (int (*)())fused7_scratch, (unsigned (*)())fused7_inplace },},
  { { (void (*)())fused8, (int (*)())fused8_scratch, (unsigned (*)())fused8_inplace },},
  { { (void (*)())fused9, (int (*)())fused9_scratch, (unsigned (*)())fused9_inplace },},
  { { (void (*)())fused10, (int (*)())fused10_scratch, (unsigned (*)())fused10_inplace },},
  { { (void (*)())fused11, (int (*)())fused11_scratch, (unsigned (*)())fused11_inplace },},
  { { (void (*)())fused12, (int (*)())fused12_scratch, (unsigned (*)())fused12_inplace },},
  { { (void (*)())fused13, (int (*)())fused13_scratch, (unsigned (*)())fused13_inplace },},
  { { (void (*)())fused14, (int (*)())fused14_scratch, (unsigned (*)())fused14_inplace },},
  { { (void (*)())fused15, (int (*)())fused15_scratch, (unsigned (*)())fused15_inplace },},
  { { (void (*)())fused16, (int (*)())fused16_scratch, (unsigned (*)())fused16_inplace },},
  { { (void (*)())fused17, (int (*)())fused17_scratch, (unsigned (*)())fused17_inplace },},
  { { (void (*)())fused18, (int (*)())fused18_scratch, (unsigned (*)())fused18_inplace },},
  { { (void (*)())fused19, (int (*)())fused19_scratch, (unsigned (*)())fused19_inplace },},
  { { (void (*)())fused20, (int (*)())fused20_scratch, (unsigned (*)())fused20_inplace },},
  { { (void (*)())fused21, (int (*)())fused21_scratch, (unsigned (*)())fused21_inplace },},
  };
/*
fused OP0#2 ($0 : SEGDES) = (DIST FLOAT @ (CONST FLOAT 100.0) $0)
fused OP1#1 ($0 : SEGDES) = (DIST FLOAT @ (CONST FLOAT 1.0) $0)
fused OP2#1 ($0 : FLOAT, $1 : INT, $2 : INT) = (* FLOAT @ $0 (/ FLOAT @ (I_TO_F @ $1) (I_TO_F @ $2)))
fused OP3#1 ($0 : FLOAT, $1 : INT) = (/ FLOAT @ $0 (I_TO_F @ $1))
fused OP4#1 ($0 : FLOAT, $1 : FLOAT, $2 : FLOAT, $3 : FLOAT, $4 : FLOAT, $5 : SEGDES, $6 : FLOAT, $7 : FLOAT, $8 : FLOAT) = (/ FLOAT
  @
  (+ FLOAT @ (LOG @ (/ FLOAT @ $0 $1))
    (* FLOAT @ (+ FLOAT @ $2 (/ FLOAT @ (* FLOAT @ $3 $4) (DIST FLOAT @ (CONST FLOAT 2.0) $5))) $6))
  (* FLOAT @ $7 (SQRT @ $8)))
fused OP5#1 ($0 : FLOAT, $1 : FLOAT, $2 : FLOAT, $3 : FLOAT, $4 : FLOAT, $5 : SEGDES, $6 : FLOAT, $7 : FLOAT, $8 : FLOAT, $9 : FLOAT, $10 : FLOAT) = (- FLOAT
  @
  (/ FLOAT @
    (+ FLOAT @ (LOG @ (/ FLOAT @ $0 $1))
      (* FLOAT @ (+ FLOAT @ $2 (/ FLOAT @ (* FLOAT @ $3 $4) (DIST FLOAT @ (CONST FLOAT 2.0) $5))) $6))
    (* FLOAT @ $7 (SQRT @ $8))) (* FLOAT @ $9 (SQRT @ $10)))
fused OP6#1 ($0 : FLOAT, $1 : FLOAT, $2 : FLOAT, $3 : FLOAT, $4 : FLOAT, $5 : FLOAT) = (- FLOAT @ (* FLOAT @ $0 $1)
  (* FLOAT @ (* FLOAT @ $2 (EXP @ (* FLOAT @ $3 $4))) $5))
fused OP7#1 ($0 : FLOAT, $1 : FLOAT, $2 : FLOAT, $3 : FLOAT, $4 : FLOAT, $5 : FLOAT) = (- FLOAT @
  (* FLOAT @ (* FLOAT @ $0 (EXP @ (* FLOAT @ $1 $2))) $3) (* FLOAT @ $4 $5))
fused OP8#1 ($0 : SEGDES, $1 : FLOAT) = (- FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.0) $0) $1)
fused OP9#1 ($0 : FLOAT, $1 : FLOAT) = (* FLOAT @ $0 (ATAN @ $1))
fused OP10#1 ($0 : FLOAT, $1 : SEGDES, $2 : FLOAT, $3 : FLOAT) = (SELECT FLOAT @
  (> FLOAT @ $0 (DIST FLOAT @ (CONST FLOAT 0.0) $1)) $2 $3)
fused OP11#1 ($0 : SEGDES, $1 : SEGDES, $2 : SEGDES, $3 : FLOAT, $4 : SEGDES, $5 : FLOAT, $6 : FLOAT) = (/ FLOAT @
  (DIST FLOAT @ (CONST FLOAT 1.0) $0)
  (+ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $1)
    (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.2316419) $2)
      (SELECT FLOAT @ (> FLOAT @ $3 (DIST FLOAT @ (CONST FLOAT 0.0) $4)) $5 $6))))
fused OP12#1 ($0 : SEGDES) = (DIST INT @ (CONST INT 3) $0)
fused OP13#1 ($0 : SEGDES) = (DIST INT @ (CONST INT 4) $0)
fused OP14#1 ($0 : SEGDES) = (DIST INT @ (CONST INT 5) $0)
fused OP15#1 ($0 : SEGDES, $1 : SEGDES, $2 : SEGDES, $3 : FLOAT, $4 : FLOAT, $5 : FLOAT, $6 : SEGDES, $7 : FLOAT, $8 : FLOAT, $9 : SEGDES, $10 : SEGDES, $11 : SEGDES, $12 : SEGDES, $13 : SEGDES, $14 : FLOAT, $15 : SEGDES, $16 : FLOAT, $17 : FLOAT, $18 : FLOAT, $19 : SEGDES, $20 : SEGDES, $21 : SEGDES, $22 : FLOAT, $23 : SEGDES, $24 : FLOAT, $25 : FLOAT, $26 : SEGDES, $27 : SEGDES, $28 : SEGDES, $29 : FLOAT, $30 : SEGDES, $31 : FLOAT, $32 : FLOAT, $33 : SEGDES, $34 : FLOAT, $35 : FLOAT, $36 : FLOAT, $37 : SEGDES, $38 : FLOAT) = (- FLOAT
  @ (DIST FLOAT @ (CONST FLOAT 1.0) $0)
  (* FLOAT @
    (* FLOAT @
      (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $1) (SQRT @ (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 2.0) $2) $3)))
      (EXP @
        (/ FLOAT @ (* FLOAT @ $4 (SELECT FLOAT @ (> FLOAT @ $5 (DIST FLOAT @ (CONST FLOAT 0.0) $6)) $7 $8))
          (DIST FLOAT @ (CONST FLOAT 2.0) $9))))
    (+ FLOAT @
      (+ FLOAT @
        (+ FLOAT @
          (+ FLOAT @
            (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.31938153) $10)
              (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $11)
                (+ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $12)
                  (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.2316419) $13)
                    (SELECT FLOAT @ (> FLOAT @ $14 (DIST FLOAT @ (CONST FLOAT 0.0) $15)) $16 $17)))))
            (* FLOAT @
              (* FLOAT @ $18
                (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $19)
                  (+ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $20)
                    (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.2316419) $21)
                      (SELECT FLOAT @ (> FLOAT @ $22 (DIST FLOAT @ (CONST FLOAT 0.0) $23)) $24 $25)))))
              (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $26)
                (+ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $27)
                  (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.2316419) $28)
                    (SELECT FLOAT @ (> FLOAT @ $29 (DIST FLOAT @ (CONST FLOAT 0.0) $30)) $31 $32))))))
          (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.781477937) $33) $34)) (* FLOAT @ $35 $36))
      (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.330274429) $37) $38))))
fused OP16#1 ($0 : FLOAT, $1 : SEGDES) = (< FLOAT @ $0 (DIST FLOAT @ (CONST FLOAT 0.0) $1))
fused OP17#1 ($0 : FLOAT, $1 : SEGDES) = (B_TO_I @ (< FLOAT @ $0 (DIST FLOAT @ (CONST FLOAT 0.0) $1)))
fused OP18#1 ($0 : SEGDES, $1 : SEGDES, $2 : SEGDES, $3 : SEGDES, $4 : FLOAT, $5 : FLOAT, $6 : FLOAT, $7 : SEGDES, $8 : FLOAT, $9 : FLOAT, $10 : SEGDES, $11 : SEGDES, $12 : SEGDES, $13 : SEGDES, $14 : SEGDES, $15 : FLOAT, $16 : SEGDES, $17 : FLOAT, $18 : FLOAT, $19 : FLOAT, $20 : SEGDES, $21 : SEGDES, $22 : SEGDES, $23 : FLOAT, $24 : SEGDES, $25 : FLOAT, $26 : FLOAT, $27 : SEGDES, $28 : SEGDES, $29 : SEGDES, $30 : FLOAT, $31 : SEGDES, $32 : FLOAT, $33 : FLOAT, $34 : SEGDES, $35 : FLOAT, $36 : FLOAT, $37 : FLOAT, $38 : SEGDES, $39 : FLOAT) = (- FLOAT
  @ (DIST FLOAT @ (CONST FLOAT 1.0) $0)
  (- FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $1)
    (* FLOAT @
      (* FLOAT @
        (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $2) (SQRT @ (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 2.0) $3) $4)))
        (EXP @
          (/ FLOAT @ (* FLOAT @ $5 (SELECT FLOAT @ (> FLOAT @ $6 (DIST FLOAT @ (CONST FLOAT 0.0) $7)) $8 $9))
            (DIST FLOAT @ (CONST FLOAT 2.0) $10))))
      (+ FLOAT @
        (+ FLOAT @
          (+ FLOAT @
            (+ FLOAT @
              (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.31938153) $11)
                (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $12)
                  (+ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $13)
                    (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.2316419) $14)
                      (SELECT FLOAT @ (> FLOAT @ $15 (DIST FLOAT @ (CONST FLOAT 0.0) $16)) $17 $18)))))
              (* FLOAT @
                (* FLOAT @ $19
                  (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $20)
                    (+ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $21)
                      (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.2316419) $22)
                        (SELECT FLOAT @ (> FLOAT @ $23 (DIST FLOAT @ (CONST FLOAT 0.0) $24)) $25 $26)))))
                (/ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $27)
                  (+ FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $28)
                    (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 0.2316419) $29)
                      (SELECT FLOAT @ (> FLOAT @ $30 (DIST FLOAT @ (CONST FLOAT 0.0) $31)) $32 $33))))))
            (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.781477937) $34) $35)) (* FLOAT @ $36 $37))
        (* FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.330274429) $38) $39)))))
fused OP19#1 ($0 : INT, $1 : INT, $2 : INT) = (= INT @ (- INT @ $0 $1) $2)
fused OP20#1 ($0 : SEGDES, $1 : FLOAT) = (- FLOAT @ (DIST FLOAT @ (CONST FLOAT 1.0) $0) $1)
fused OP21#1 ($0 : INT, $1 : FLOAT) = (* FLOAT @ (EXP @ (I_TO_F @ $0)) (LOG @ $1))
*/
