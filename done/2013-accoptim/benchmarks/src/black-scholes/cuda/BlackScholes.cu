#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */


#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>        // helper functions CUDA error checking and initialization

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(
    float *h_CallResult,
    float *h_PutResult,
    float *h_StockPrice,
    float *h_OptionStrike,
    float *h_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
// const int OPT_N = 4000000;
// const int  NUM_ITERATIONS = 512;

#define OPT_N           4000000
#define NUM_ITERATIONS  512


// const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // Start logs
    printf("[%s] - Starting...\n", argv[0]);

    //'h_' prefix - CPU (host) memory space
    float
    //Results calculated by CPU for reference
    *h_CallResultCPU,
    *h_PutResultCPU,
    //CPU copy of GPU results
    *h_CallResultGPU,
    *h_PutResultGPU,
    //CPU instance of input data
    *h_StockPrice,
    *h_OptionStrike,
    *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
    //Results calculated by GPU
    *d_CallResult,
    *d_PutResult,
    //GPU instance of input data
    *d_StockPrice,
    *d_OptionStrike,
    *d_OptionYears;

//    double delta, ref, sum_delta, sum_ref, max_delta, L1norm;
    double gpuTime;

    StopWatchInterface *hTimer = NULL;
    int i;

    findCudaDevice(argc, (const char **)argv);

    // determine how many options to process
    int opt_n = OPT_N;
    if (checkCmdLineFlag(argc, (const char**) argv, "options"))
    {
        opt_n   = getCmdLineArgumentInt(argc, (const char**)argv, "options");
        if (opt_n < 1) {
            printf("Error: \"number of options\" specified %d is invalid\n", opt_n);
            exit(EXIT_FAILURE);
        }
    }
    const int opt_sz = opt_n * sizeof(float);

    // how many benchmarking iterations
    int num_iterations = NUM_ITERATIONS;
    if (checkCmdLineFlag(argc, (const char**) argv, "samples"))
    {
        num_iterations = getCmdLineArgumentInt(argc, (const char**)argv, "samples");
        if (opt_n < 1) {
            printf("Error: \"number of benchmark samples\" specified %d is invalid\n", num_iterations);
            exit(EXIT_FAILURE);
        }
    }

    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory for options.\n");
    h_CallResultCPU = (float *)malloc(opt_sz);
    h_PutResultCPU  = (float *)malloc(opt_sz);
    h_CallResultGPU = (float *)malloc(opt_sz);
    h_PutResultGPU  = (float *)malloc(opt_sz);
    h_StockPrice    = (float *)malloc(opt_sz);
    h_OptionStrike  = (float *)malloc(opt_sz);
    h_OptionYears   = (float *)malloc(opt_sz);

    printf("...allocating GPU memory for options.\n");
//    checkCudaErrors(hipMalloc((void **)&d_CallResult,   opt_sz));
//    checkCudaErrors(hipMalloc((void **)&d_PutResult,    opt_sz));
    checkCudaErrors(hipMalloc((void **)&d_StockPrice,   opt_sz));
    checkCudaErrors(hipMalloc((void **)&d_OptionStrike, opt_sz));
    checkCudaErrors(hipMalloc((void **)&d_OptionYears,  opt_sz));

    printf("...generating input data in CPU mem.\n");
    srand(5347);

    //Generate options set
    for (i = 0; i < opt_n; i++)
    {
        h_CallResultCPU[i] = 0.0f;
        h_PutResultCPU[i]  = -1.0f;
        h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
        h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
        h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
    }

    printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing
    checkCudaErrors(hipMemcpy(d_StockPrice,  h_StockPrice,   opt_sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionStrike, h_OptionStrike,  opt_sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionYears,  h_OptionYears,   opt_sz, hipMemcpyHostToDevice));
    printf("Data init done.\n\n");


    printf("Executing Black-Scholes GPU kernel (%i iterations)...\n", num_iterations);
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for (i = 0; i < num_iterations; i++)
    {
        checkCudaErrors(hipMalloc((void **)&d_CallResult, opt_sz));    // TLM
        checkCudaErrors(hipMalloc((void **)&d_PutResult,  opt_sz));    // TLM

        BlackScholesGPU<<<480, 128>>>(
            d_CallResult,
            d_PutResult,
            d_StockPrice,
            d_OptionStrike,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            opt_n
        );
        getLastCudaError("BlackScholesGPU() execution failed\n");

        checkCudaErrors(hipFree(d_PutResult));         // TLM
        checkCudaErrors(hipFree(d_CallResult));        // TLM
    }

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    gpuTime = sdkGetTimerValue(&hTimer) / num_iterations;

    //Both call and put is calculated
    printf("Options count             : %i\n",      opt_n);
    printf("BlackScholesGPU() time    : %f msec\n", gpuTime);
    printf("Effective memory bandwidth: %f GB/s\n", ((double)(5 * opt_n * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
    printf("Gigaoptions per second    : %f\n\n",    ((double)(2 * opt_n) * 1E-9) / (gpuTime * 1E-3));

    printf("BlackScholes, Throughput = %.4f GOptions/s, Time = %.5f s, Size = %u options, NumDevsUsed = %u, Workgroup = %u\n",
           (((double)(2.0 * opt_n) * 1.0E-9) / (gpuTime * 1.0E-3)), gpuTime*1e-3, (2 * opt_n), 1, 128);

#if 0
    printf("\nReading back GPU results...\n");
    //Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_CallResultGPU, d_CallResult, opt_sz, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_PutResultGPU,  d_PutResult,  opt_sz, hipMemcpyDeviceToHost));


    printf("Checking the results...\n");
    printf("...running CPU calculations.\n\n");
    //Calculate options values on CPU
    BlackScholesCPU(
        h_CallResultCPU,
        h_PutResultCPU,
        h_StockPrice,
        h_OptionStrike,
        h_OptionYears,
        RISKFREE,
        VOLATILITY,
        opt_n
    );

    printf("Comparing the results...\n");
    //Calculate max absolute difference and L1 distance
    //between CPU and GPU results
    sum_delta = 0;
    sum_ref   = 0;
    max_delta = 0;

    for (i = 0; i < opt_n; i++)
    {
        ref   = h_CallResultCPU[i];
        delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);

        if (delta > max_delta)
        {
            max_delta = delta;
        }

        sum_delta += delta;
        sum_ref   += fabs(ref);
    }

    L1norm = sum_delta / sum_ref;
    printf("L1 norm: %E\n", L1norm);
    printf("Max absolute error: %E\n\n", max_delta);
#endif

    printf("Shutting down...\n");
    printf("...releasing GPU memory.\n");
    checkCudaErrors(hipFree(d_OptionYears));
    checkCudaErrors(hipFree(d_OptionStrike));
    checkCudaErrors(hipFree(d_StockPrice));
//    checkCudaErrors(hipFree(d_PutResult));
//    checkCudaErrors(hipFree(d_CallResult));

    printf("...releasing CPU memory.\n");
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_PutResultGPU);
    free(h_CallResultGPU);
    free(h_PutResultCPU);
    free(h_CallResultCPU);
    sdkDeleteTimer(&hTimer);

    hipDeviceReset();
    printf("Shutdown done.\n");

#if 0
    printf("\n[BlackScholes] - Test Summary\n");

    if (L1norm > 1e-6)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    exit(EXIT_SUCCESS);
#endif
}
